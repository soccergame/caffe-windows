#include "hip/hip_runtime.h"
#include <cfloat>
#include "caffe/util/math_functions.hpp"
#include "caffe/layers/ordinal_regression_loss_layer.hpp"

namespace caffe {

template<typename Dtype>
__global__ void kernel_ordreg_forward(const int k, const int n,
    const Dtype* x, Dtype* y, const Dtype* label, const Dtype* weight, Dtype* loss) {
  CUDA_KERNEL_LOOP(idx, n) {
    const int sample_idx = idx / k;
    const int label_idx = idx % k;
    const int offset = 2*idx;
    const int this_label = static_cast<int>(label[sample_idx]);
    const Dtype this_weight = weight[label_idx];
    const Dtype* x_data = x + offset;
    Dtype* y_data = y + offset;
    Dtype* loss_data = loss + offset;
    Dtype max_input = max(x_data[0], x_data[1]);
    y_data[0] = exp(x_data[0] - max_input);
    y_data[1] = exp(x_data[1] - max_input);
    Dtype sum = y_data[0] + y_data[1];
    y_data[0] /= sum;
    y_data[1] /= sum;
    if (label_idx < this_label) {
      loss_data[0] = 0;
      loss_data[1] = -log(max(y[1], Dtype(FLT_MIN)));
    }
    else {
      loss_data[0] = -log(max(y[0], Dtype(FLT_MIN)));
      loss_data[1] = 0;
    }
    loss_data[0] *= this_weight;
    loss_data[1] *= this_weight;
  }
}

template<typename Dtype>
void OrdinalRegressionLossLayer<Dtype>::Forward_gpu(
    const vector<Blob<Dtype>*>& bottom, const vector<Blob<Dtype>*>& top) {
  const int n = bottom[0]->shape(0);
  const int nthread = n * k_;
  const Dtype* x = bottom[0]->gpu_data();
  const Dtype* label = bottom[1]->gpu_data();
  const Dtype* weight = weight_.gpu_data();
  Dtype* y = prob_.mutable_gpu_data();
  Dtype* loss_data = bottom[0]->mutable_gpu_diff();  // reuse
  kernel_ordreg_forward<Dtype><<<CAFFE_GET_BLOCKS(nthread),
      CAFFE_CUDA_NUM_THREADS>>>(k_, nthread, x, y, label, weight, loss_data);
  Dtype loss;
  caffe_gpu_asum(bottom[0]->count(), loss_data, &loss);
  top[0]->mutable_cpu_data()[0] = loss / n;
}

template<typename Dtype>
__global__ void kernel_ordreg_backward(const int k, const int n,
    Dtype* dx, const Dtype* label, const Dtype* weight) {
  CUDA_KERNEL_LOOP(idx, n) {
    const int sample_idx = idx / k;
    const int label_idx = idx % k;
    const int offset = 2*idx;
    const int this_label = static_cast<int>(label[sample_idx]);
    const Dtype this_weight = weight[label_idx];
    Dtype* dx_data = dx + offset;
    if (label_idx < this_label) {
      dx_data[1] -= 1;
    }
    else {
      dx_data[0] -= 1;
    }
    dx_data[0] *= this_weight;
    dx_data[1] *= this_weight;
  }
}

template<typename Dtype>
void OrdinalRegressionLossLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
    const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (propagate_down[1]) {
    LOG(FATAL) << this->type() << " Layer cannot backpropagate to label inputs.";
  }
  if (propagate_down[0]) {
    const int n = bottom[0]->shape(0);
    const int nthread = n * k_;
    const Dtype* prob_data = prob_.gpu_data();
    const Dtype* label_data = bottom[1]->gpu_data();
    const Dtype* weight_data = weight_.gpu_data();
    Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
    caffe_gpu_memcpy(prob_.count() * sizeof(Dtype), prob_data, bottom_diff);
    kernel_ordreg_backward<Dtype><<<CAFFE_GET_BLOCKS(nthread),
        CAFFE_CUDA_NUM_THREADS>>>(k_, nthread, bottom_diff, label_data, weight_data);
    const Dtype scale = 1.0 / n;
    caffe_gpu_scal<Dtype>(prob_.count(), scale, bottom_diff);
  }
}

INSTANTIATE_LAYER_GPU_FUNCS(OrdinalRegressionLossLayer);

}  // namespace caffe
